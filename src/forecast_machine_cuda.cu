#include "hip/hip_runtime.h"
#include "forecast_machine_cuda.h"

// __device__ void find_nearest_neighbors_cuda(const vec& dist, std::vector<size_t> &return_nearest_neighbors, size_t nn, std::vector<size_t> which_lib, double epsilon)
__device__ void find_nearest_neighbors_cuda_test()
{
    // printf("Test");
    return;
}

// __device__ void adjust_lib(const size_t curr_pred)
// {
//     // clear out lib indices we don't want from which_lib
//     if(exclusion_radius >= 0)
//     {
//         auto f = [&](const size_t curr_lib) {
//             return (curr_lib == curr_pred) || ((time[curr_lib] >= (time[curr_pred] - exclusion_radius)) && (time[curr_lib] <= (time[curr_pred] + exclusion_radius)));
//         };
//         which_lib.erase(std::remove_if(which_lib.begin(), which_lib.end(), f), which_lib.end());
//     }
//     else
//     {
//         which_lib.erase(std::remove(which_lib.begin(), which_lib.end(), curr_pred), which_lib.end());
//     }
//     return;
// }

__global__ void simplex_prediction_cuda(const size_t start, const size_t end, size_t *which_pred, size_t *which_lib)
{
    // int index = blockIdx.x * blockDim.x + threadIdx.x;
    // int stride = blockDim.x * gridDim.x;
    // // printf("Thread: %d, Block: %d\n", threadIdx.x, blockIdx.x);
    // find_nearest_neighbors_cuda_test();
    // for (int i = index; i < n; i += stride)
    //     y[i] = x[i] + y[i];

    printf("GPU\n");

    printf("Start: %lu\n", start);
    printf("End: %lu\n", end);

    for(size_t k = start; k < end; ++k)
    {
        size_t curr_pred = which_pred[k];

        printf("%lu\n", curr_pred);
        // printf("%lu\n", which_lib[k]);

        // find nearest neighbors
        // if(CROSS_VALIDATION)
        // {
            // size_t temp_lib = which_lib;
            // adjust_lib(curr_pred);
        //     // find_nearest_neighbors_cuda<<<1, 1>>>(distances[curr_pred], nearest_neighbors, nn, which_lib, epsilon);

        //     find_nearest_neighbors_cuda(distances[curr_pred], nearest_neighbors);
            // which_lib = temp_lib;
        // }
        // else
        // {

        //     find_nearest_neighbors(distances[curr_pred], nearest_neighbors);
        // }
    }
}

void call_cuda(const size_t start, const size_t end, std::vector<size_t> which_pred, std::vector<size_t> which_lib, size_t **which_lib_adjusted)
{
    size_t *which_pred_array;
    size_t *which_lib_array;
    size_t *which_lib_adjusted_array;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&which_pred_array, which_pred.size()*sizeof(size_t));
    hipMallocManaged(&which_lib_array, which_lib.size()*sizeof(size_t));
    hipMallocManaged(&which_lib_array, which_lib.size()*which_lib.size()*sizeof(size_t));

    std::copy(which_pred.begin(), which_pred.end(), which_pred_array);
    std::copy(which_lib.begin(), which_lib.end(), which_lib_array);

    // Run kernel on 1M elements on the GPU
    // int blockSize = 256;
    // int numBlocks = (N + blockSize - 1) / blockSize;
    simplex_prediction_cuda<<<1,1>>>(start, end, which_pred_array, which_lib_array);
    hipError_t error = hipGetLastError();    
    if (error != hipSuccess)
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(error));

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Free memory
    hipFree(which_pred_array);
    hipFree(which_lib_array);

    return;
}

std::vector<size_t> find_nearest_neighbors_cuda(const vec& dist, size_t nn, std::vector<size_t> which_lib, double epsilon) 
{
    if(nn < 1)
    {
        return sort_indices_cuda(dist, which_lib);
    }
    // else
    std::vector<size_t> neighbors;
    std::vector<size_t> nearest_neighbors;
    double curr_distance;

    if(nn > log(double(which_lib.size())))
    {
        // printf("True");
        neighbors = sort_indices_cuda(dist, which_lib);
        std::vector<size_t>::iterator curr_lib;

        // find nearest neighbors
        for(curr_lib = neighbors.begin(); curr_lib != neighbors.end(); ++curr_lib)
        {
            nearest_neighbors.push_back(*curr_lib);
            if(nearest_neighbors.size() >= nn)
                break;
        }
        if(curr_lib == neighbors.end())
            return nearest_neighbors;

        double tie_distance = dist[nearest_neighbors.back()];

        // check for ties
        for(++curr_lib; curr_lib != neighbors.end(); ++curr_lib)
        {
            if(dist[*curr_lib] > tie_distance) // distance is bigger
                break;
            nearest_neighbors.push_back(*curr_lib); // add to nearest neighbors
        }
    }
    else
    {
        // printf("False");
        size_t i;
        for(auto curr_lib: which_lib)
        {
            // distance to current neighbor under examination
            curr_distance = dist[curr_lib];
            
            // We want to include the current neighbor:
            //   if haven't populated neighbors vector, or
            //   if current neighbor is nearer than farthest away neighbor
            if(nearest_neighbors.size() < nn || 
               curr_distance <= dist[nearest_neighbors.back()])
            {
                // find the correct place to insert the current neighbor
                i = nearest_neighbors.size();
                while((i > 0) && (curr_distance < dist[nearest_neighbors[i-1]]))
                {
                    i--;
                }
                nearest_neighbors.insert(nearest_neighbors.begin()+i, curr_lib);

                // if we've added too many neighbors and there isn't a tie, then
                // pop off the farthest neighbor
                while((nearest_neighbors.size() > nn) &&
                   (dist[nearest_neighbors[nn-1]] < dist[nearest_neighbors.back()]))
                {
                    nearest_neighbors.pop_back();
                }
            }
        }
    }

    // filter for max_distance
    if(epsilon >= 0)
    {
        for(auto neighbor_iter = nearest_neighbors.begin(); neighbor_iter != nearest_neighbors.end(); ++neighbor_iter)
        {
            if(dist[*neighbor_iter] > epsilon)
            {
                nearest_neighbors.erase(neighbor_iter, nearest_neighbors.end());
                break;
            }
        }
    }

    return nearest_neighbors;
}

std::vector<size_t> sort_indices_cuda(const vec& v, std::vector<size_t> idx)
{
    sort(idx.begin(), idx.end(),
         [&v](size_t i1, size_t i2) {return v[i1] < v[i2];});
    return idx;
}